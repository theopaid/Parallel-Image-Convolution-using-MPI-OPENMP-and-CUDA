#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "cuda_convolution.h"
#include "functions.h"
#include "hip/hip_runtime.h"
#include ""

// Each thread takes care of one element of vector 'src'
__global__ void kernelConvolutionforGrey(uint8_t *src, uint8_t *dst, int img_width, int img_height) {
    int i, j, k, l;
    // Filter initialization
    int my_filter[3][3] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};
    // get position
    size_t x = blockIdx.x*blockDim.x + threadIdx.x;
    size_t y = blockIdx.y*blockDim.y + threadIdx.y;
    // convolute
    if (0 < x && x < img_height-1 && 0 < y && y < img_width-1) {
        float afterFilter = 0;
        for (i = x-1, k = 0 ; i <= x+1 ; i++, k++)
            for (j = y-1, l = 0 ; j <= y+1 ; j++, l++)
                afterFilter += src[img_width * i + j] * my_filter[k][l] / 16.0;
        dst[img_width * x + y] = afterFilter;
    }
}

__global__ void kernelConvolutionforRGB(uint8_t *src, uint8_t *dst, int img_width, int img_height) {
    int i, j, k, l;
    // Filter inialization
    int my_filter[3][3] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};
    // get position
    size_t x = blockIdx.x*blockDim.x + threadIdx.x;
    size_t y = blockIdx.y*blockDim.y + threadIdx.y;
    // convolute
    if (0 < x && x < img_height-1 && 0 < y && y < 3*img_width-3) {
        float afterFilterRED = 0, afterFIlterGREEN = 0, afterFilterBLUE = 0;
        for (i = x-1, k = 0 ; i <= x+1 ; i++, k++) {
            for (j = (y*3)-3, l = 0 ; j <= (y*3)+3 ; j+=3, l++) {
                afterFilterRED += src[(img_width*3) * i + j]* my_filter[k][l] /16.0;
                afterFIlterGREEN += src[(img_width*3) * i + j+1] * my_filter[k][l] /16.0;
                afterFilterBLUE += src[(img_width*3) * i + j+2] * my_filter[k][l] /16.0;
            }
        }
        dst[img_width*3 * x + (y*3)] = afterFilterRED;
        dst[img_width*3 * x + (y*3)+1] = afterFIlterGREEN;
        dst[img_width*3 * x + (y*3)+2] = afterFilterBLUE;
    }
}

extern "C" void GPU_convolution(uint8_t *src, int img_width, int img_height, int repetitions, color_t img_type)
{
    // Vectors
    uint8_t *d_src, *d_dst, *tmp;
    size_t bytes = (img_type == GREY) ? img_height * img_width : img_height * img_width*3;

    // Allocate memory for each vector on GPU
    CUDA_SAFE_CALL( hipMalloc(&d_src, bytes * sizeof(uint8_t)) );
    CUDA_SAFE_CALL( hipMalloc(&d_dst, bytes * sizeof(uint8_t)) );
 
    // Copy host vectors to device memory
    CUDA_SAFE_CALL( hipMemcpy(d_src, src, bytes, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemset(d_dst, 0, bytes) );

    int t;
    const int blockSize = 16;
    // Convolute "repetition" times
    for (t = 0 ; t < repetitions ; t++) {
        
        if (img_type == GREY) {
            // Specify layout of Grid and Blocks
            int gridX = FRACTION_CEILING(img_height, blockSize);
            int gridY = FRACTION_CEILING(img_width, blockSize);
            dim3 block(blockSize, blockSize);
            dim3 grid(gridX, gridY);
            kernelConvolutionforGrey<<<grid, block>>>(d_src, d_dst, img_width, img_height);
        } else if (img_type == RGB) {
            int gridX = FRACTION_CEILING(img_height, blockSize);
            int gridY = FRACTION_CEILING(img_width*3, blockSize);
            dim3 block(blockSize, blockSize);
            dim3 grid(gridX, gridY);
            kernelConvolutionforRGB<<<grid, block>>>(d_src, d_dst, img_width, img_height);
        }

        // swap arrays
        tmp = d_src;
        d_src = d_dst;
        d_dst = tmp;
    }

    CUDA_SAFE_CALL( hipGetLastError() );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    
    // Copy array back to host
    if (repetitions%2 == 0) {
        CUDA_SAFE_CALL( hipMemcpy(src, d_src, bytes, hipMemcpyDeviceToHost) );
    } else {
           CUDA_SAFE_CALL( hipMemcpy(src, d_dst, bytes, hipMemcpyDeviceToHost) );
       }

    // Release device memory
    CUDA_SAFE_CALL( hipFree(d_src) );
    CUDA_SAFE_CALL( hipFree(d_dst) );
}